#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_world() {
    printf("Hello world!\n");
}

int main() {
    hello_world<<<10, 2>>>();
    hipDeviceSynchronize();
    return 0;
}